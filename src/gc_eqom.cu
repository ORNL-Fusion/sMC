#include "hip/hip_runtime.h"
#include "eqdsk.hpp"
#include "rk_gc_particle.hpp"
#include "constants.hpp"
#include <iostream>
#include "interp.hpp"

using namespace std;

// Calculate vGC given position, mu and vPar.
__host__ __device__
Crk vGC ( const REAL dt, const Crk &p0, const REAL mu, 
            Ceqdsk &eqdsk, const interpSpans &spans, int &err ) {

	Crk vGC;

	// get background data(s) at particle location
	
	interpIndex index;
	index = get_index ( p0.r, p0.z,	spans );

	if(index.stat) {

        /*
	    cout << "\t" << __FILE__  << "\tREAL i: "<<index.i << endl;
	    cout << "\t" << __FILE__  << "\tREAL j: "<<index.j << endl;
	    cout << "\t" << __FILE__  << "\tREAL r: "<< p0.r << endl;
	    cout << "\t" << __FILE__  << "\tREAL z: "<< p0.z << endl;
        */

        err++;
        return vGC;
    }

    REAL bmag = bilinear_interp ( index, eqdsk.bmag );

    REAL b_r = bilinear_interp ( index, eqdsk.br );
    REAL b_p = bilinear_interp ( index, eqdsk.bp );
    REAL b_z = bilinear_interp ( index, eqdsk.bz );

    REAL bCurv_r = bilinear_interp ( index, eqdsk.bCurvature_r );
    REAL bCurv_p = bilinear_interp ( index, eqdsk.bCurvature_p );
    REAL bCurv_z = bilinear_interp ( index, eqdsk.bCurvature_z );

    REAL bGrad_r = bilinear_interp ( index, eqdsk.bGradient_r );
    REAL bGrad_p = bilinear_interp ( index, eqdsk.bGradient_p );
    REAL bGrad_z = bilinear_interp ( index, eqdsk.bGradient_z );

    REAL bDotGradB = bilinear_interp ( index, eqdsk.bDotGradB );

	REAL unitb_r = b_r / bmag;
	REAL unitb_p = b_p / bmag;
	REAL unitb_z = b_z / bmag;

	// vPer
	REAL vPer = sqrt ( 2.0 * mu * bmag / _mi );
	// dvPar_dt
	REAL dvPar_dt = -mu / _mi * bDotGradB;
	// Here vGC is a dvGC and so vGC.vPar is really a dvPar.
	// I'm just using the Crk class as containers for x/dx and v/dv quantities.
	vGC.vPar = dvPar_dt; 
	REAL vPar = p0.vPar;

	// vGC
	vGC.r = vPar * unitb_r + pow(vPer,2) * bGrad_r + pow(vPar,2) * bCurv_r;
	vGC.p = vPar * unitb_p + pow(vPer,2) * bGrad_p + pow(vPar,2) * bCurv_p;
	vGC.z = vPar * unitb_z + pow(vPer,2) * bGrad_z + pow(vPar,2) * bCurv_z;

	return vGC;
}


