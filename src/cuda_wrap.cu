#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include "particle.hpp"
#include "eqdsk.hpp"
#include "constants.hpp"
#include "cuda_wrap.h"
#include "cuPrintf.cu"

using namespace std;

/*
__device__ int cu_get_index 
	( const REAL rIn, const REAL zIn, cu_interpIndex &index ) const {

	index.j = (rIn - r.front()) / ( r.back() - r.front() ) * (r.size()-1.0);
	index.i = (zIn - z.front()) / ( z.back() - z.front() ) * (z.size()-1.0);

	index.i1 = floor(index.i);
	index.i2 = ceil(index.i);
	index.j1 = floor(index.j);
	index.j2 = ceil(index.j);

    // Check if particle is off grid	
    if( index.i1<0 || index.i2>=(z.size()-1) ||
        index.j1<0 || index.j2>=(r.size()-1) ) {
        return 1;
    }

	return 0;
}
*/
__global__ void testKernelA(int val) {
	cuPrintf("Value is: %d\n", val);
}

__global__ void check2Dcpy ( REAL *data2D, 
				size_t pitch, const unsigned int nRow, const unsigned int nCol ) {

	for (int r=0;r<nRow;++r) {
			REAL *row = (REAL*)((char*)data2D + r*pitch);
			for (int c=0;c<nCol;++c) {
					REAL element = row[c];
                    cuPrintf("%i %i %f\n", r, c, element);
			}
	}
}

__global__ void check1Dcpy ( REAL *data1D, const unsigned int n ) {

	for(int i=0;i<n;++i) {
			cuPrintf("%i %f\n", i, data1D[i]);
	}
}

int copy_particles_to_device (vector<Cgc_particle> &H_particles) {


    cout << "First CUDA call :)" << endl;

    cout << "\tCopying particle list from HOST -> DEVICE ... ";
    thrust::device_vector<Cgc_particle> D_particles ( H_particles.begin(), H_particles.end() );
    cout << "DONE" << endl;

    //cout << "\tCopying particle list DEVICE -> HOST ... ";
    //thrust::host_vector<Cgc_particle> H_tmp ( D_particles.begin(), D_particles.end() );
    //cout << "DONE" << endl;

    //float **D_test2D;
    //hipMalloc( (void**) &D_test2D, eqdsk.nRow * eqdsk.nCol * sizeof(float *));

	return 0;
}

cu_ptr_pitch copy_2D_to_device 
( boost::multi_array<REAL,2> &data2D, const unsigned int M, const unsigned int N ) {

    cu_ptr_pitch out;
	size_t size = N * sizeof(REAL);

	hipMallocPitch ( (void**)&out.ptr, &out.pitch, size, M );
	hipMemcpy2D ( out.ptr, out.pitch, &data2D[0][0], 
					size, size, M, hipMemcpyHostToDevice );

	return out;
}

REAL* copy_1D_to_device 
( std::vector<REAL> &h_data1D, const unsigned int n ) {

	REAL *d_data1D;
	size_t size = n * sizeof(REAL);
	hipMalloc ( (void**)&d_data1D, size );
	hipMemcpy ( d_data1D, &h_data1D[0], size, hipMemcpyHostToDevice);

	return d_data1D;
}

int cu_test_cuda ( const cu_ptrs &d_ptrs, const int nRow, const int nCol ) {

    cudaPrintfInit();

    cout << "Launching testKernel ..." << endl;

	check1Dcpy<<<1,1>>>( d_ptrs.z, nRow );
	check2Dcpy<<<1,1>>>( d_ptrs.bmag.ptr, d_ptrs.bmag.pitch, nRow, nCol );

    cudaPrintfDisplay (stdout, true);
    cudaPrintfEnd();

    return 0;
}

